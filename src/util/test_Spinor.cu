#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2025 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "Spinor.h"
#include <vector>

/*
  Multiply Spinors by Jones matrix
*/
__global__ void transform 
(
  const Jones<double>& jones,
  Spinor<double>* spinors,
  unsigned ndat
)
{
  const unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= ndat)
    return;

  spinors[idx] = jones * spinors[idx];
}

/*
  Compute array of instantaneous Stokes parameters from an array of spinors
*/
__global__ void compute_stokes 
(
  Vector<4,double>* stokes,
  Spinor<double>* spinors,
  unsigned ndat
)
{
  const unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= ndat)
    return;

  compute_stokes(stokes[idx], spinors[idx]);
}

void launch ()
{
  Jones<double> jones;
  std::vector<Spinor<double>> spinors (1024);
  std::vector<Vector<4,double>> stokes (1024);

  dim3 threads (512);
  dim3 blocks (spinors.size() / threads.x);
  if (spinors.size() % threads.x)
    blocks.x ++;

  transform<<<blocks,threads>>> (jones, spinors.data(), spinors.size());
  compute_stokes<<<blocks,threads>>> (stokes.data(), spinors.data(), spinors.size());
}
