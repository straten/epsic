#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2025 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "cuSpinor.h"
#include "Stokes.h"
#include "Quaternion.h"
#include "Pauli.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>

#include <iostream>
#include <ctime>

int test_cuSpinor()
{
  size_t nsamp = 1024*1024;
  thrust::device_vector<Spinor<double>> e_field (nsamp);

  NormalSpinor generator;
  generator.set_seed(time(NULL));

  Stokes<double> mean (1.0, 0.0, 0.2, 0.7);
  Quaternion<double,Hermitian> root = sqrt (natural(mean));
  generator.set_polarizer(convert(root));

  thrust::generate(e_field.begin(), e_field.end(), generator);

  thrust::host_vector<Spinor<double>> on_host = e_field;

  for (int i = 0; i < 10; ++i)
  {
    std::cout << "  " << i << ": " << on_host[i] << std::endl;
  }
  std::cout << std::endl;

  return 0;
}
