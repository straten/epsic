#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2025 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "cuSpinor.h"
#include "Stokes.h"
#include "Quaternion.h"
#include "Pauli.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/transform.h>

#include <iostream>
#include <ctime>

template<typename T>
class instantaneous_stokes
{
  __prefix__ Stokes<T> operator() (const Spinor<T>& e)
  {
    Vector<4,T> tmp;
    compute_stokes (tmp, e);
    return tmp;
  }
};

int test_cuSpinor()
{
  NormalSpinor generator;
  generator.set_seed(time(NULL));

  Stokes<double> mean (1.0, 0.0, 0.2, 0.7);
  Quaternion<double,Hermitian> root = 0.5 * sqrt (natural(mean));
  generator.set_polarizer(convert(root));

  size_t nsamp = 1024*1024;
  thrust::device_vector<Spinor<double>> e_field (nsamp);
  thrust::generate(e_field.begin(), e_field.end(), generator);

  thrust::device_vector<Stokes<double>> stokes (nsamp);
  thrust::transform(e_field.begin(), e_field.end(), stokes.begin(), instantaneous_stokes<double>());

  Stokes<double> sum = thrust::reduce(stokes.begin(), stokes.end());

  std::cout << "mean=" << sum/nsamp << std::endl;

  return 0;
}
