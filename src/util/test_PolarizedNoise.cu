#include "hip/hip_runtime.h"
//-*-C++-*-

/***************************************************************************
 *
 *   Copyright (C) 2025 by Willem van Straten
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include "PolarizedNoise.h"
#include "Stokes.h"
#include "Quaternion.h"
#include "Pauli.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/transform.h>

#include <iostream>
#include <ctime>

using namespace std;

int test_PolarizedNoise()
{
  PolarizedNoise<double> generator;
  generator.set_seed(time(NULL));

  Stokes<double> mean (1.0, 0.0, 0.2, 0.7);
  Quaternion<double,Hermitian> root = 0.5 * sqrt (natural(mean));
  generator.set_polarizer(convert(root));

  size_t nsamp = 1024*1024;

  {
    cerr << "generating on device using thrust" << endl;

    thrust::device_vector<Spinor<double>> e_field (nsamp);
    thrust::generate(e_field.begin(), e_field.end(), generator);

    cerr << "transforming on device using thrust" << endl;

    thrust::device_vector<Stokes<double>> stokes (nsamp);
    thrust::transform(e_field.begin(), e_field.end(), stokes.begin(), instantaneous_stokes<double>());

    cerr << "summing on device using thrust" << endl;

    Stokes<double> sum = thrust::reduce(stokes.begin(), stokes.end());

    cerr << "thrust device mean=" << sum/nsamp << endl << endl;
  }

  cerr << "generating on host using thrust" << endl;

  thrust::host_vector<Spinor<double>> e_field (nsamp);
  thrust::generate(e_field.begin(), e_field.end(), generator);

  cerr << "transforming on host using thrust" << endl;

  thrust::host_vector<Stokes<double>> stokes (nsamp);
  thrust::transform(e_field.begin(), e_field.end(), stokes.begin(), instantaneous_stokes<double>());

  cerr << "summing on host using thrust" << endl;

  Stokes<double> sum = thrust::reduce(stokes.begin(), stokes.end());

  cerr << "thrust host mean=" << sum/nsamp << endl;

  return 0;
}
